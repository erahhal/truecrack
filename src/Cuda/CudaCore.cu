#include "hip/hip_runtime.h"
/*
 * Copyright (C)  2011  Luca Vaccaro
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "Tcdefs.h"
#include "Volumes.h"
#include <stdio.h>
#include <memory.h>
#include "Crypto.h"
#include "CudaCore.cuh"
#include "CudaPkcs5.cuh"
#include "CudaXts.cuh"



/* The max number of block grid; number of max parallel gpu blocks. */
int blockGridSizeMax;

/* The number of the current block grid; number of current parallel gpu blocks. */
int blockGridSizeCurrent;

/* Pointer of structures to pass to Cuda Kernel. */
unsigned char *dev_salt, *dev_blockPwd, *dev_header;
int *dev_blockPwd_init, *dev_blockPwd_length;
short int *dev_result;


int getMultiprocessorCount (void){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.multiProcessorCount;
}

//#define RESIDENTTHREADS		1536
//#define NUMBLOCKS		12
#define NUMTHREADSXBLOCK	256

static void HandleError( hipError_t err, const char *file,  int line ) {
        if (err != hipSuccess) {
                printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
                exit( EXIT_FAILURE );
        }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__device__ void computePwd (uint64_t number, uint64_t maxcombination, int charsetlength, unsigned char *charset, int wordlength, unsigned char *word){
    unsigned short i=0;
    if (number>=maxcombination) return;
    for (i=0;i<wordlength;i++)
        word[i]=0;
    i=0;
    while(number>0){
        word[i]=number%charsetlength;
        number=(number-word[i])/charsetlength;
        i++;
    }
    for (i=0;i<wordlength;i++)
	word[i]=charset[word[i]];
	
}
__device__ void cuda_Pbkdf2_charset ( unsigned char *salt, unsigned char *pwd, int pwd_len, unsigned char *headerkey, int numBlock) {
	SupportPkcs5 support;
	SupportPkcs5 *sup;
	sup = &support;
	
	//INCLUDE: void derive_u_ripemd160 (char *pwd, int pwd_len, char *salt, int salt_len, int iterations, char *u, int b)

	int c, i;	
	int b=numBlock;
	unsigned char *u=headerkey+RIPEMD160_DIGESTSIZE*b;

	// iteration 1 
	memset (sup->ccounter, 0, 4);
	sup->ccounter[3] = (char) b;
	memcpy (sup->cinit, salt, SALT_LENGTH);	// salt 
	memcpy (&sup->cinit[SALT_LENGTH],sup->ccounter, 4);	// big-endian block number 
	
	cuda_hmac_ripemd160 (pwd, pwd_len, sup->cinit, SALT_LENGTH + 4, sup->cj, sup);
	memcpy (u, sup->cj, RIPEMD160_DIGESTSIZE);
	
	//remaining iterations 
	for (c = 1; c < ITERATIONS; c++)
	{
		cuda_hmac_ripemd160 (pwd, pwd_len, sup->cj, RIPEMD160_DIGESTSIZE, sup->ck,sup);
		for (i = 0; i < RIPEMD160_DIGESTSIZE; i++)
		{
			u[i] ^= sup->ck[i];
			sup->cj[i] = sup->ck[i];
		}
	}
}

 __global__ void cuda_Kernel_charset (
    	unsigned char *salt,
    	unsigned char *headerEncrypted,
    	unsigned short int charset_length,
    	unsigned char *charset,
    	unsigned short int password_length,
    	uint64_t maxcombination,
    	 short int *result)
 {
	uint64_t numData = blockIdx.x*blockDim.x+threadIdx.x;
	__align__(8) unsigned char headerkey[192];
	__align__(8) unsigned char headerDecrypted[512];
	__align__(8) unsigned char pwd[8];

	//__device__ void computePwd (int number, int maxcombination, int charsetlength, unsigned char *charset, int wordlength, unsigned char *word){
	computePwd(numData,maxcombination,charset_length,charset,password_length,pwd);
	pwd[password_length]='\0';
	
	//__device__ void cuda_Pbkdf2_charset_ ( unsigned char *salt, unsigned char *pwd, int pwd_len, unsigned char *headerkey) {
	cuda_Pbkdf2_charset_ ( salt, pwd, password_length, headerkey);

	int value=cuda_Xts (headerEncrypted, headerkey, headerDecrypted);
	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}

__global__ void cuda_Kernel ( unsigned char *salt, unsigned char *headerEncrypted, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int max) {
	int value;
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;

	if (numData>=max) return;

	// Array of unsigned char in the shared memory
	__align__(8) unsigned char headerkey[192];
	__align__(8) unsigned char headerDecrypted[512];

	// Calculate the hash header key
	int i=0;
	for (i=0;i<10;i++)
		cuda_Pbkdf2 (salt, blockPwd, blockPwd_init, blockPwd_length, headerkey, numData,i);


	// Decrypt the header and compare the key
	value=cuda_Xts (headerEncrypted, headerkey,headerDecrypted);

	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}

/*
__global__ void cuda_Kernel ( unsigned char *salt, unsigned char *headerEncrypted, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int max) {
	int value;
	int numData=blockIdx.x;//threadIdx.x;

	if (numData>=max) return;

	// Array of unsigned char in the shared memory
	__shared__ __align__(8) unsigned char headerkey[192];
	__shared__ __align__(8) unsigned char headerDecrypted[512];

	// Calculate the hash header key
	int i=0;
	//for (i=0;i<10;i++)
		cuda_Pbkdf2 (salt, blockPwd, blockPwd_init, blockPwd_length, headerkey, blockIdx.x,threadIdx.x);

	__syncthreads();

if(threadIdx.x==0){
	// Decrypt the header and compare the key
	value=cuda_Xts (headerEncrypted, headerkey,headerDecrypted);

	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}
}
*/
/*
void cuda_Core ( int block_currentsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result) {	
	hipStream_t stream[2];
	for (int i = 0; i < 2; ++i)
	    hipStreamCreate(&stream[i]);

	int lengthpwd=0;
	for (int j=0;j<block_currentsize;j++) {
		lengthpwd+=blockPwd_length[j];
	}

	hipMalloc ( &dev_result, block_currentsize * sizeof(short int)) ;

	unsigned char *host_blockPwd;
	int *host_blockPwd_init, *host_blockPwd_length;
	short int *host_result;
	hipHostAlloc( (void**) &host_blockPwd, lengthpwd *sizeof(unsigned char),hipHostMallocDefault );
	hipHostAlloc( (void**) &host_blockPwd_init, block_currentsize *sizeof(int),hipHostMallocDefault );
	hipHostAlloc( (void**) &host_blockPwd_length, block_currentsize *sizeof(int),hipHostMallocDefault );
	hipHostAlloc( (void**) &host_result, block_currentsize *sizeof(short int),hipHostMallocDefault );

	for (int i=0;i<block_currentsize;i++){
		host_blockPwd_init[i]=blockPwd_init[i];
		host_blockPwd_length[i]=blockPwd_length[i];
		host_result[i]=result[i];
	}
	for (int i=0;i<lengthpwd;i++)
		host_blockPwd[i]=blockPwd[i];

	int i;
	for (i=0;i<block_currentsize;i++) {
		result[i]=NODEFINED;
	}
	
#define STREAM 1
int sizePwd=block_currentsize;
int sizeStream=sizePwd/STREAM;
int lenghtPwdInStream[STREAM];


	int numBlocks=sizeStream/NUMTHREADSXBLOCK+1;
	int numThread=NUMTHREADSXBLOCK;
	if (sizeStream<NUMTHREADSXBLOCK)
		numThread=sizeStream;

	for (i = 0; i < STREAM; ++i) {
	    lenghtPwdInStream[i]=0;
	    for (int j=i*sizeStream;j<sizeStream;j++) {
		lenghtPwdInStream[i]+=blockPwd_length[j];
	    }
	}
int j;	
	for (i = 0; i < STREAM; ++i) {
	    hipMemcpyAsync(dev_blockPwd_init+i*sizeStream, host_blockPwd_init+i*sizeStream, sizeStream* sizeof(int) , hipMemcpyHostToDevice,stream[i]);
	    hipMemcpyAsync(dev_blockPwd_length+i*sizeStream, host_blockPwd_length+i*sizeStream, sizeStream * sizeof(int) , hipMemcpyHostToDevice,stream[i]) ;
	    hipMemcpyAsync(dev_result+i*sizeStream, host_result+i*sizeStream, sizeStream * sizeof(short int) , hipMemcpyHostToDevice,stream[i]) ;
	    hipMemcpyAsync(dev_blockPwd+i*lenghtPwdInStream[i-1], host_blockPwd+i*lenghtPwdInStream[i-1], lenghtPwdInStream[i] * sizeof(unsigned char), hipMemcpyHostToDevice,stream[i]) ;
	    
	    printf("start: %s\n",host_blockPwd+i*lenghtPwdInStream[i-1]);
	    printf("%d : %d - %d \n",sizeStream,numBlocks,numThread);

	    
	    cuda_Kernel<<<numBlocks,numThread,0,stream[i]>>>(dev_salt, dev_header, dev_blockPwd+i*lenghtPwdInStream[i-1], dev_blockPwd_init+i*sizeStream, dev_blockPwd_length+i*sizeStream, dev_result+i*sizeStream,sizeStream);
	    
	    
	    hipError_t err=hipMemcpyAsync(host_result+i*sizeStream, dev_result+i*sizeStream,sizeStream * sizeof(short int) , hipMemcpyDeviceToHost,stream[i]) ;
	    if (err!=hipSuccess){
		  printf("->%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
	    }
   //  hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size,       size, hipMemcpyDeviceToHost, stream[i]);

	    hipMemcpyAsync(inputDevPtr + i * size, hostPtr + i * size,
			    size, hipMemcpyHostToDevice, stream[i]);
	    
	    MyKernel <<<100, 512, 0, stream[i]>>>
		  (outputDevPtr + i * size, inputDevPtr + i * size, size);
	    hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size,
			    size, hipMemcpyDeviceToHost, stream[i]);
	}

	
	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);

	for (i=0;i<STREAM;i++)
	    for (j=0;j<sizeStream;j++)
		printf("%d -> %c\n",j+i*sizeStream,result[j+i*sizeStream]);
	
	for (int i = 0; i < STREAM; ++i)
	    hipStreamDestroy(stream[i]);
}
*/

void cuda_Core_dictionary ( int block_currentsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result) {

	int lengthpwd=0;
	for (int j=0;j<block_currentsize;j++) {
		lengthpwd+=blockPwd_length[j];
		result[j]=0;
	}
	hipMalloc ( &dev_result, block_currentsize * sizeof(short int)) ;
	hipMemcpy(dev_blockPwd, blockPwd, lengthpwd * sizeof(unsigned char) , hipMemcpyHostToDevice) ;
	hipMemcpy(dev_blockPwd_init, blockPwd_init, block_currentsize * sizeof(int) , hipMemcpyHostToDevice);
	hipMemcpy(dev_blockPwd_length, blockPwd_length, block_currentsize * sizeof(int) , hipMemcpyHostToDevice) ;
	hipMemcpy(dev_result, result, block_currentsize * sizeof(short int) , hipMemcpyHostToDevice) ;

	int numBlocks=block_currentsize/NUMTHREADSXBLOCK+1;
	int numThread=NUMTHREADSXBLOCK;
	if (block_currentsize<NUMTHREADSXBLOCK)
		numThread=block_currentsize;

	cuda_Kernel<<<numBlocks,numThread>>>(dev_salt, dev_header, dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_result,block_currentsize);

	hipError_t err=hipMemcpy(result, dev_result,block_currentsize* sizeof(short int) , hipMemcpyDeviceToHost) ;
	if (err!=hipSuccess){
		printf("->%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
	}
	hipFree(dev_result);
}

void cuda_Core_charset ( unsigned short int charset_length, unsigned char *charset, unsigned short int password_length, short int *result) 
{
	uint64_t maxcombination=1;
	for (int i=0;i<password_length;i++)
		maxcombination*=charset_length;

	unsigned char *dev_charset = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_charset, charset_length*sizeof(unsigned char)));
	HANDLE_ERROR(hipMalloc ( (void **)&dev_result, maxcombination * sizeof(short int))) ;
	HANDLE_ERROR(hipMemcpy(dev_charset, charset, charset_length*sizeof(unsigned char), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_result, result, maxcombination*sizeof(short int), hipMemcpyHostToDevice));
     
	uint64_t numBlocks=maxcombination/NUMTHREADSXBLOCK+1;
	int numThread=NUMTHREADSXBLOCK;
	if (maxcombination<NUMTHREADSXBLOCK)
		numThread=maxcombination;

	cuda_Kernel_charset<<<numBlocks,numThread>>>(dev_salt, dev_header, charset_length, dev_charset, password_length, maxcombination,dev_result);
	
	// Copy the device result vector in device memory to the host result vector in host memory.
	HANDLE_ERROR( hipMemcpy(result, dev_result, maxcombination*sizeof(short int), hipMemcpyDeviceToHost));
	
	HANDLE_ERROR(hipFree(dev_charset));;
	HANDLE_ERROR(hipFree(dev_result));;
}

   

void cuda_Init (int block_maxsize, unsigned char *salt, unsigned char *header) {
	blockGridSizeMax=block_maxsize;

	hipMalloc ( (void **)&dev_blockPwd, blockGridSizeMax * PASSWORD_MAXSIZE * sizeof(unsigned char)) ;
	hipMalloc ( (void **)&dev_blockPwd_init, blockGridSizeMax * sizeof(int)) ;
	hipMalloc ( (void **)&dev_blockPwd_length, blockGridSizeMax * sizeof(int)) ;
	hipMalloc ( (void **)&dev_salt, SALT_LENGTH * sizeof(unsigned char)) ;
	hipMalloc ( (void **)&dev_header, TC_VOLUME_HEADER_EFFECTIVE_SIZE * sizeof(unsigned char)) ;

	hipMemcpy(dev_salt, salt, SALT_LENGTH * sizeof(unsigned char) , hipMemcpyHostToDevice) ;
	hipMemcpy(dev_header, header, TC_VOLUME_HEADER_EFFECTIVE_SIZE * sizeof(unsigned char) , hipMemcpyHostToDevice) ;

}

void cuda_Free(void) {
	hipFree(dev_salt);
	hipFree(dev_blockPwd);
	hipFree(dev_blockPwd_init);
	hipFree(dev_blockPwd_length);
	hipFree(dev_header);
}
